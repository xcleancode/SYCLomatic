// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/cusolver_helper_function_2 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolver_helper_function_2/cusolver_helper_function_2.dp.cpp --match-full-lines %s

//CHECK:#include <sycl/sycl.hpp>
//CHECK-NEXT:#include <dpct/dpct.hpp>
//CHECK-NEXT:#include <dpct/lapack_utils.hpp>

#include <hip/hip_runtime.h>
#include "hipsolver.h"

int foo1() {
  hipsolverHandle_t handle;
  float ** a_s_ptrs;
  double ** a_d_ptrs;
  float2 ** a_c_ptrs;
  double2 ** a_z_ptrs;
  int *infoArray;
  hipsolverStatus_t status;

  //CHECK:status = dpct::lapack::potrf_batch(*handle, oneapi::mkl::uplo::upper, 3, a_s_ptrs, 3, infoArray, 2);
  //CHECK:status = dpct::lapack::potrf_batch(*handle, oneapi::mkl::uplo::upper, 3, a_d_ptrs, 3, infoArray, 2);
  //CHECK:status = dpct::lapack::potrf_batch(*handle, oneapi::mkl::uplo::upper, 3, a_c_ptrs, 3, infoArray, 2);
  //CHECK:status = dpct::lapack::potrf_batch(*handle, oneapi::mkl::uplo::upper, 3, a_z_ptrs, 3, infoArray, 2);
  status = hipsolverDnSpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, a_s_ptrs, 3, infoArray, 2);
  status = hipsolverDnDpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, a_d_ptrs, 3, infoArray, 2);
  status = hipsolverDnCpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, a_c_ptrs, 3, infoArray, 2);
  status = hipsolverDnZpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, a_z_ptrs, 3, infoArray, 2);

  return 0;
}

int foo2() {
  hipsolverHandle_t handle;
  float ** a_s_ptrs, ** b_s_ptrs;
  double ** a_d_ptrs, ** b_d_ptrs;
  float2 ** a_c_ptrs, ** b_c_ptrs;
  double2 ** a_z_ptrs, ** b_z_ptrs;
  int *infoArray;
  hipsolverStatus_t status;

  //CHECK:status = dpct::lapack::potrs_batch(*handle, oneapi::mkl::uplo::upper, 3, 1, a_s_ptrs, 3, b_s_ptrs, 3, infoArray, 2);
  //CHECK:status = dpct::lapack::potrs_batch(*handle, oneapi::mkl::uplo::upper, 3, 1, a_d_ptrs, 3, b_d_ptrs, 3, infoArray, 2);
  //CHECK:status = dpct::lapack::potrs_batch(*handle, oneapi::mkl::uplo::upper, 3, 1, a_c_ptrs, 3, b_c_ptrs, 3, infoArray, 2);
  //CHECK:status = dpct::lapack::potrs_batch(*handle, oneapi::mkl::uplo::upper, 3, 1, a_z_ptrs, 3, b_z_ptrs, 3, infoArray, 2);
  status = hipsolverDnSpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, a_s_ptrs, 3, b_s_ptrs, 3, infoArray, 2);
  status = hipsolverDnDpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, a_d_ptrs, 3, b_d_ptrs, 3, infoArray, 2);
  status = hipsolverDnCpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, a_c_ptrs, 3, b_c_ptrs, 3, infoArray, 2);
  status = hipsolverDnZpotrsBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, 1, a_z_ptrs, 3, b_z_ptrs, 3, infoArray, 2);

  return 0;
}

void foo3() {
  hipsolverHandle_t handle;
  void *a_s, *a_d, *a_c, *a_z;
  void *s_s, *s_d, *s_c, *s_z;
  void *u_s, *u_d, *u_c, *u_z;
  void *vt_s, *vt_d, *vt_c, *vt_z;
  int device_ws_size_s;
  int device_ws_size_d;
  int device_ws_size_c;
  int device_ws_size_z;

  //CHECK:int gesvdjinfo;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusolverDnCreateGesvdjInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  hipsolverGesvdjInfo_t gesvdjinfo;
  hipsolverDnCreateGesvdjInfo(&gesvdjinfo);

  //CHECK:dpct::lapack::gesvd_scratchpad_size(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::real_float, 2, dpct::library_data_t::real_float, 2, dpct::library_data_t::real_float, 2, &device_ws_size_s);
  //CHECK-NEXT:dpct::lapack::gesvd_scratchpad_size(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::real_double, 2, dpct::library_data_t::real_double, 2, dpct::library_data_t::real_double, 2, &device_ws_size_d);
  //CHECK-NEXT:dpct::lapack::gesvd_scratchpad_size(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::complex_float, 2, dpct::library_data_t::complex_float, 2, dpct::library_data_t::complex_float, 2, &device_ws_size_c);
  //CHECK-NEXT:dpct::lapack::gesvd_scratchpad_size(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::complex_double, 2, dpct::library_data_t::complex_double, 2, dpct::library_data_t::complex_double, 2, &device_ws_size_z);
  hipsolverDnSgesvdj_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (float*)a_s, 2, (float*)s_s, (float*)u_s, 2, (float*)vt_s, 2, &device_ws_size_s, gesvdjinfo);
  hipsolverDnDgesvdj_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (double*)a_d, 2, (double*)s_d, (double*)u_d, 2, (double*)vt_d, 2, &device_ws_size_d, gesvdjinfo);
  hipsolverDnCgesvdj_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (float2*)a_c, 2, (float*)s_c, (float2*)u_c, 2, (float2*)vt_c, 2, &device_ws_size_c, gesvdjinfo);
  hipsolverDnZgesvdj_bufferSize(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (double2*)a_z, 2, (double*)s_z, (double2*)u_z, 2, (double2*)vt_z, 2, &device_ws_size_z, gesvdjinfo);

  void* device_ws_s;
  void* device_ws_d;
  void* device_ws_c;
  void* device_ws_z;
  int *info;

  //CHECK:dpct::lapack::gesvd(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::real_float, (float*)a_s, 2, dpct::library_data_t::real_float, (float*)s_s, dpct::library_data_t::real_float, (float*)u_s, 2, dpct::library_data_t::real_float, (float*)vt_s, 2, (float*)device_ws_s, device_ws_size_s, info);
  //CHECK-NEXT:dpct::lapack::gesvd(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::real_double, (double*)a_d, 2, dpct::library_data_t::real_double, (double*)s_d, dpct::library_data_t::real_double, (double*)u_d, 2, dpct::library_data_t::real_double, (double*)vt_d, 2, (double*)device_ws_d, device_ws_size_d, info);
  //CHECK-NEXT:dpct::lapack::gesvd(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::complex_float, (sycl::float2*)a_c, 2, dpct::library_data_t::real_float, (float*)s_c, dpct::library_data_t::complex_float, (sycl::float2*)u_c, 2, dpct::library_data_t::complex_float, (sycl::float2*)vt_c, 2, (sycl::float2*)device_ws_c, device_ws_size_c, info);
  //CHECK-NEXT:dpct::lapack::gesvd(*handle, oneapi::mkl::job::vec, 0, 2, 2, dpct::library_data_t::complex_double, (sycl::double2*)a_z, 2, dpct::library_data_t::real_double, (double*)s_z, dpct::library_data_t::complex_double, (sycl::double2*)u_z, 2, dpct::library_data_t::complex_double, (sycl::double2*)vt_z, 2, (sycl::double2*)device_ws_z, device_ws_size_z, info);
  hipsolverDnSgesvdj(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (float*)a_s, 2, (float*)s_s, (float*)u_s, 2, (float*)vt_s, 2, (float*)device_ws_s, device_ws_size_s, info, gesvdjinfo);
  hipsolverDnDgesvdj(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (double*)a_d, 2, (double*)s_d, (double*)u_d, 2, (double*)vt_d, 2, (double*)device_ws_d, device_ws_size_d, info, gesvdjinfo);
  hipsolverDnCgesvdj(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (float2*)a_c, 2, (float*)s_c, (float2*)u_c, 2, (float2*)vt_c, 2, (float2*)device_ws_c, device_ws_size_c, info, gesvdjinfo);
  hipsolverDnZgesvdj(handle, HIPSOLVER_EIG_MODE_VECTOR, 0, 2, 2, (double2*)a_z, 2, (double*)s_z, (double2*)u_z, 2, (double2*)vt_z, 2, (double2*)device_ws_z, device_ws_size_z, info, gesvdjinfo);

  //CHECK:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusolverDnDestroyGesvdjInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  hipsolverDnDestroyGesvdjInfo(gesvdjinfo);
}
