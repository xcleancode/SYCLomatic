
#include <hip/hip_runtime.h>
// RUN: rm -rf %T/Util/api_test26_out
// RUN: dpct --use-custom-helper=api --out-root=%T/Util/api_test26_out \
// RUN:      --cuda-include-path="%cuda-path/include" \
// RUN:      %s -- -x cuda -ptx
// RUN: grep "kernel_wrapper:" %T/Util/api_test26_out/MainSourceFiles.yaml \
// RUN: | python -c "len(input().splitlines()) == 1"

// TEST_FEATURE: Util_kernel_wrapper

extern "C" __global__ void kernel(int *x, short y) {}
