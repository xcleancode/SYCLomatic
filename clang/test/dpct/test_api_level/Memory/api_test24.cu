
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test24_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test24_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test24_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test24_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test24_out

// CHECK: 48
// TEST_FEATURE: Memory_device_memory_get_access
// TEST_FEATURE: Memory_device_memory_init
// TEST_FEATURE: Memory_dpct_accessor

__device__ float c[16][16];

__global__ void kernel() {
  c[0][0] = 1.0f;
}

int main() {
  kernel<<<1, 1>>>();
  return 0;
}
