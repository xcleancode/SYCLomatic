
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test2_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test2_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test2_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test2_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test2_out

// CHECK: 32
// TEST_FEATURE: Memory_dpct_malloc_3d
// TEST_FEATURE: Memory_dpct_malloc_2d

int main() {
  hipExtent extent = make_hipExtent(1, 1, 1);
  hipPitchedPtr p3;
  hipMalloc3D(&p3, extent);
  return 0;
}
