
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test37_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test37_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test37_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test37_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test37_out

// CHECK: 36
// TEST_FEATURE: Memory_dpct_memcpy
// TEST_FEATURE: Memory_dpct_memcpy_2d
// TEST_FEATURE: Memory_dpct_memcpy_3d

int main() {
  hipMemcpy3DParms parms;
  hipMemcpy3D(&parms);
  return 0;
}
