// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test14_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test14_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test14_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test14_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test14_out

// CHECK: 34
// TEST_FEATURE: LapackUtils_potrs


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  float* b_s;
  hipsolverHandle_t handle;
  size_t device_ws_size_s;
  size_t host_ws_size_s;
  hipsolverDnParams_t params;
  void* device_ws_s;
  void* host_ws_s;
  int *info;

  hipsolverDnXpotrs(handle, params, HIPBLAS_FILL_MODE_LOWER, 3, 1, HIP_R_32F, a_s, 3, HIP_R_32F, b_s, 3, info);
  return 0;
}
