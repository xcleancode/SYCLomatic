// RUN: dpct --format-range=none -in-root %S -out-root %T/explicit_namespace_default %S/explicit_namespace_default.cu --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/explicit_namespace_default/explicit_namespace_default.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <cmath>
#include <hip/hip_runtime.h>


__device__ float4 fun() {
  float4 a, b, c;
  // CHECK: sycl::fma(a.x(), b.x(), c.x());
  __fmaf_rn(a.x, b.x, c.x);
  // CHECK: return sycl::float4(sycl::fma(a.x(), b.x(), c.x()), sycl::fma(a.y(), b.y(), c.y()), sycl::fma(a.z(), b.z(), c.z()), sycl::fma(a.w(), b.w(), c.w()));
  return make_float4(__fmaf_rd(a.x, b.x, c.x), __fmaf_rz(a.y, b.y, c.y), __fmaf_rn(a.z, b.z, c.z), __fmaf_rn(a.w, b.w, c.w));
}


__global__ void kernel() {

}

void foo() {
  // CHECK:   dpct::get_default_queue().parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, ceil(2.3)), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {
  // CHECK-NEXT:           kernel();
  // CHECK-NEXT:         });
  kernel<<< ceil(2.3), 1 >>>();
}

int main() {

}